#include <stdio.h>
#include "hip/hip_runtime.h"

// Define matrix width
#define N 100
#define BLOCK_DIM 32
#define SIGMA 20.0
// Define tile size
#define TILE_WIDTH 2

// Non shared version
__global__ void computeMatrix(float *dVectorA, float *dVectorB, float *dVectorC, int length, float sigma)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int tid = length * y + x;
	float tmp = 0;

    if (x < length && y < length)
	{
        tmp = dVectorA[tid] - dVectorB[tid];
		tmp = (tmp*tmp)/(2*(sigma*sigma));
		dVectorC[tid] = exp(-tmp);
	}
}

// Shared version doesn't work
__global__ void computeMatrixShared(float *dVectorA, float *dVectorB, float *dVectorC, int length, float sigma)
{
	__shared__ float Ads[TILE_WIDTH][TILE_WIDTH];
	__shared__ float Bds[TILE_WIDTH][TILE_WIDTH];

	float tmp = 0;
	unsigned int col = TILE_WIDTH*blockIdx.x + threadIdx.x;
	unsigned int row = TILE_WIDTH*blockIdx.y + threadIdx.y;
	
	for(int m = 0; m < length/TILE_WIDTH; m++)
	{
		Ads[threadIdx.y][threadIdx.x] = dVectorA[row * length +(m * TILE_WIDTH + threadIdx.x)];
		Bds[threadIdx.y][threadIdx.x] = dVectorB[(m*TILE_WIDTH + threadIdx.y) * length + col];
		// Synchronize all threads
		__syncthreads();
		for(int k = 0; k < TILE_WIDTH; k++)
		{
			tmp = Ads[threadIdx.x][k] + Bds[k][threadIdx.y];
			tmp = (tmp*tmp)/(2*(sigma*sigma));
			dVectorC[row * length +  col] = exp(-tmp);
		}

		// Synchronize all threads
		__syncthreads();
	}
}

int main()
{
    hipSetDevice(0);

	int totalLength = N * N;
    float hVectorA[totalLength];
    float hVectorB[totalLength];
    float hVectorC[totalLength];
    float *dVectorA = NULL;
    float *dVectorB = NULL; 
    float *dVectorC = NULL; 
	// Fill arrays
    for (int i = 0; i < totalLength; i++)
    {
        hVectorA[i] = 2*i;
        hVectorB[i] = 1*i;
    }
	int size = sizeof(float) * totalLength;
	// Transfert A and B to device
    hipMalloc((void**) &dVectorA, size);
    hipMalloc((void**) &dVectorB, size);
    hipMalloc((void**) &dVectorC, size);
    hipMemcpy(dVectorA, hVectorA, size, hipMemcpyHostToDevice);
    hipMemcpy(dVectorB, hVectorB, size, hipMemcpyHostToDevice);

	// -- Non shared version
	// -- Grid mapping
    dim3 blocks((totalLength + BLOCK_DIM - 1) / BLOCK_DIM);
    dim3 threads(BLOCK_DIM);

	// -- Kernel invocation code
    computeMatrix<<<blocks, threads>>>(dVectorA, dVectorB, dVectorC, N, SIGMA);

	// -- Shared version
	// -- Grid mapping
    //dim3 dimGrid ( N/TILE_WIDTH , N/TILE_WIDTH ,1 ) ;
	//dim3 dimBlock( TILE_WIDTH, TILE_WIDTH, 1 ) ;

	// -- Kernel invocation code
    //computeMatrixShared<<<dimGrid, dimBlock>>>(dVectorA, dVectorB, dVectorC, N, (float)SIGMA);

	// Transfert C from device to host
    hipMemcpy(hVectorC, dVectorC, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < totalLength; i++)
        printf("%0.1f\t", hVectorC[i]);
    printf("\n");
	
	// Free memories
	hipFree(dVectorA);
	hipFree(dVectorB);
	hipFree(dVectorC);

	return 0;
}
